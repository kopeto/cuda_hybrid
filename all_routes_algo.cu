#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
//#include <hip/hip_runtime.h>

#include <cstdio>
#include <stdio.h>
#include <stdlib.h>

#include "Graph.hpp"

#define CUDA_CALL(x) do { hipError_t error = x; if((x)!=hipSuccess) { \
    printf("Cuda Error at %s:%d -- ",__FILE__,__LINE__);                \
    printf("%s\nAbort.\n",hipGetErrorString(error));                   \
    exit(0);                                                            \
    }} while(0)


__global__ void INITIALIZE
(
	uint32_t * const d_distances, 
	bool * const d_frontier,
	const uint32_t source,
	const uint32_t n_vertex
)
{
	const uint32_t node = blockDim.x * blockIdx.x + threadIdx.x;
	if(node<n_vertex)
	{
		d_frontier[node] = false;
		d_distances[node] = 0;
		if(node == source)
		{
			d_frontier[node] = true;
		}
	}
}

__global__ void GET_DISTANCES
(
	const uint32_t * const d_adjacency_offsets, 
	const uint32_t * const d_adjacency_list,
	uint32_t * const d_distances, 
	bool * const d_frontier,
	const int round,
	const uint32_t n_vertex, 
	const uint32_t n_edges
)
{
	//uint32_t max_threads = blockDim.x * gridDim.x;
	uint32_t node = blockDim.x * blockIdx.x + threadIdx.x;

	if(node<n_vertex && d_frontier[node])
	{
		d_frontier[node] = 0;
		int offset = d_adjacency_offsets[node];
		
		while(offset < d_adjacency_offsets[node+1])
		{
			int adj_node = d_adjacency_list[offset];
			d_distances[adj_node] |= (1 << (round));
			d_frontier[adj_node] = 1;
			++offset;
		}
	}
}


void Graph::get_all_distances(const uint32_t MAX_ROUNDS)
{
    // GPU pointers location prparation:

    //Graph
	uint32_t    *d_adjacency_offsets= NULL;
	uint32_t    *d_adjacency_list= NULL;

    // Algorithm
	bool        *d_frontier = NULL;
    
	// Results:
	uint32_t	*d_distances= NULL;

	distances = (uint32_t **)malloc(n_vertex * sizeof(uint32_t *));

	CUDA_CALL(hipMalloc((void **)&d_adjacency_offsets, n_vertex*sizeof(uint32_t)));
    CUDA_CALL(hipMalloc((void **)&d_adjacency_list, n_edges*sizeof(uint32_t)));
    CUDA_CALL(hipMalloc((void **)&d_frontier,      n_vertex*sizeof(bool)));
    CUDA_CALL(hipMalloc((void **)&d_distances,    n_vertex*sizeof(uint32_t)));
	
	// Copy graph into GPU:
	CUDA_CALL(hipMemcpy(d_adjacency_offsets, adjacency_offsets, n_vertex*sizeof(uint32_t), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(d_adjacency_list, adjacency_list, n_edges*sizeof(uint32_t), hipMemcpyHostToDevice));

	dim3 BLOCK(WARP_SIZE);
    dim3 GRID((n_vertex+BLOCK.x-1)/BLOCK.x);

	int source_counter = 0;
	do{ // Get all distances from each source node
		// Set kernell beginning parameters
		
		INITIALIZE<<<GRID,BLOCK>>> 
		(
			d_distances, 
			d_frontier,
			source_counter,
			n_vertex
		);

		// KERNEL
		int round = 0;
			
		while(round<MAX_ROUNDS)
		{
			GET_DISTANCES<<<GRID,BLOCK>>>
			(
				d_adjacency_offsets, 
				d_adjacency_list,
				d_distances, 
				d_frontier,
				round,
				n_vertex, 
				n_edges
			);
			//CUDA_CALL(hipDeviceSynchronize());
			round ++;
		}

		// Get results
		uint32_t *_distances = (uint32_t *)malloc(n_vertex*sizeof(uint32_t));
		CUDA_CALL(hipMemcpy(_distances, d_distances, n_vertex*sizeof(uint32_t), hipMemcpyDeviceToHost));
		*(distances+source_counter) = _distances;

		++source_counter;

	} while(source_counter < n_vertex);

	hipFree(d_frontier);
	hipFree(d_adjacency_offsets);
	hipFree(d_adjacency_list);

    return;
};

void Graph::get_all_distances_from_single_source(const uint32_t source, const uint32_t MAX_ROUNDS)
{
    // GPU pointers location prparation:

    //Graph
	uint32_t    *d_adjacency_offsets= NULL;
	uint32_t    *d_adjacency_list= NULL;

    // Algorithm
	bool        *d_frontier = NULL;
    
	// Results:
	uint32_t	*d_distances= NULL;

	distances = (uint32_t **)malloc(n_vertex * sizeof(uint32_t *));

	CUDA_CALL(hipMalloc((void **)&d_adjacency_offsets, n_vertex*sizeof(uint32_t)));
    CUDA_CALL(hipMalloc((void **)&d_adjacency_list, n_edges*sizeof(uint32_t)));
    CUDA_CALL(hipMalloc((void **)&d_frontier,      n_vertex*sizeof(bool)));
    CUDA_CALL(hipMalloc((void **)&d_distances,    n_vertex*sizeof(uint32_t)));

	
	// Copy graph into GPU:
	CUDA_CALL(hipMemcpy(d_adjacency_offsets, adjacency_offsets, n_vertex*sizeof(uint32_t), hipMemcpyHostToDevice));
	CUDA_CALL(hipMemcpy(d_adjacency_list, adjacency_list, n_edges*sizeof(uint32_t), hipMemcpyHostToDevice));

	dim3 BLOCK(WARP_SIZE);
    dim3 GRID((n_vertex+BLOCK.x-1)/BLOCK.x);

	// Get all distances from each source node
	// Set kernell beginning parameters
	INITIALIZE<<<GRID,BLOCK>>> 
	(
		d_distances, 
		d_frontier,
		source,
		n_vertex
	);

	// KERNEL
	int round = 0;
		
	while(round<MAX_ROUNDS)
	{
		GET_DISTANCES<<<GRID,BLOCK>>>
		(
			d_adjacency_offsets, 
			d_adjacency_list,
			d_distances, 
			d_frontier,
			round,
			n_vertex, 
			n_edges
		);
		//CUDA_CALL(hipDeviceSynchronize());
		round ++;
	}

	// Get results
	uint32_t *_distances = (uint32_t *)malloc(n_vertex*sizeof(uint32_t));
	CUDA_CALL(hipMemcpy(_distances, d_distances, n_vertex*sizeof(uint32_t), hipMemcpyDeviceToHost));
	*(distances+source) = _distances;

	hipFree(d_frontier);
	hipFree(d_adjacency_offsets);
	hipFree(d_adjacency_list);

    return;
};

